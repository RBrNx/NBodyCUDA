//GPU code goes here
#include <hip/hip_runtime.h>
#include <cuda_occupancy.h>
#include <malloc.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

//__global__ void initValues(int *input, int *output, int size){
//	int idx = threadIdx.x + blockIdx.x * blockDim.x;
//
//	if (idx < size) { output[idx] = input[idx] * 2; }
//}
//
//int main(){
//	const int size = 1000000;
//	srand(time(NULL));
//
//	int blockSize;
//	int minGridSize;
//	int gridSize;
//
//	int* h_Array = (int*) malloc(size * sizeof(int));
//	int* h_testArray = (int*)malloc(size * sizeof(int));
//
//	int* d_InputArray; cudaMalloc((void**)&d_InputArray, size * sizeof(int));
//	int* d_OutputArray; cudaMalloc((void**)&d_OutputArray, size * sizeof(int));
//
//	//Test
//	for (int i = 0; i < size; i++){
//		h_Array[i] = i;
//		h_testArray[i] = h_Array[i] * 2;
//	}
//
//	cudaMemcpy(d_InputArray, h_Array, size * sizeof(int), cudaMemcpyHostToDevice);
//
//	cudaOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, initValues, 0, size);
//
//	gridSize = (size + blockSize - 1) / blockSize;
//
//	initValues <<<gridSize, blockSize >>>(d_InputArray, d_OutputArray, size);
//
//	cudaMemcpy(h_Array, d_OutputArray, size*sizeof(int), cudaMemcpyDeviceToHost);
//
//	for (int i = 0; i < size; i++){
//		if (h_Array[i] != h_testArray[i]){
//			printf("Error at %i ! Host = %i, Device = %i \n", i, h_testArray[i], h_Array[i]);
//		}
//	}
//
//	int random = rand() % size;
//	printf("Random Number: %i, Host Value at %i, Device Value at %i \n", random, h_testArray[random], h_Array[random]);
//
//	printf("Test Passed \n");
//
//}

/*
Running without arguments is equivalent to 1000 iterations with the
5 celestial objects declared in the golden_bodies array.

$ nbody.exe 1000 5

The output of this shows the energy before and after the simulation,
and should be:

-0.169075164
-0.169087605
*/

#include <chrono>
#include <cmath>
#include <cstdlib>
#include <iomanip>
#include <iostream>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

using type = double;

const type pi{ 3.141592653589793 };
const type solar_mass{ 4 * pi * pi };
const type days_per_year{ 365.24 };

int blockSize;
int minGridSize;
int gridSize;

template <typename T>
struct planet {
	T x, y, z;
	T vx, vy, vz;
	T mass;
};

template <typename T>
void advance(int nbodies, planet<T> *bodies)
{
	int i, j;

	for (i = 0; i < nbodies; ++i) {
		planet<T> &b = bodies[i];

		for (j = i + 1; j < nbodies; j++) {
			planet<T> &b2 = bodies[j];
			T dx = b.x - b2.x;
			T dy = b.y - b2.y;
			T dz = b.z - b2.z;
			T inv_distance = 1.0 / sqrt(dx * dx + dy * dy + dz * dz);
			T mag = inv_distance * inv_distance * inv_distance;
			b.vx -= dx * b2.mass * mag;
			b.vy -= dy * b2.mass * mag;
			b.vz -= dz * b2.mass * mag;
			b2.vx += dx * b.mass  * mag;
			b2.vy += dy * b.mass  * mag;
			b2.vz += dz * b.mass  * mag;
		}
	}

	for (i = 0; i < nbodies; ++i) {
		planet<T> &b = bodies[i];
		b.x += b.vx;
		b.y += b.vy;
		b.z += b.vz;
	}
}

template <typename T>
T energy(int nbodies, planet<T> *bodies) {
	T e = 0.0;

	//GPU
	for (int i = 0; i < nbodies; ++i) {
		planet<T> &b = bodies[i];
		e += 0.5 * b.mass * (b.vx * b.vx + b.vy * b.vy + b.vz * b.vz);

		for (int j = i + 1; j < nbodies; j++) {
			planet<T> &b2 = bodies[j];
			T dx = b.x - b2.x;
			T dy = b.y - b2.y;
			T dz = b.z - b2.z;
			T distance = sqrt(dx * dx + dy * dy + dz * dz);
			e -= (b.mass * b2.mass) / distance;
		}
	}
	return e;
}

template <typename T>
void offset_momentum(int nbodies, planet<T> *bodies) {
	T px = 0.0, py = 0.0, pz = 0.0;

	//GPU
	for (int i = 0; i < nbodies; ++i) {
		px += bodies[i].vx * bodies[i].mass;
		py += bodies[i].vy * bodies[i].mass;
		pz += bodies[i].vz * bodies[i].mass;
	}

	bodies[0].vx = -px / solar_mass;
	bodies[0].vy = -py / solar_mass;
	bodies[0].vz = -pz / solar_mass;
}

struct planet<type> golden_bodies[5] = {
	{                               /* sun */
		0, 0, 0, 0, 0, 0, solar_mass
	},

	{                               /* jupiter */
		4.84143144246472090e+00,
		-1.16032004402742839e+00,
		-1.03622044471123109e-01,
		1.66007664274403694e-03 * days_per_year,
		7.69901118419740425e-03 * days_per_year,
		-6.90460016972063023e-05 * days_per_year,
		9.54791938424326609e-04 * solar_mass
	},

	{                               /* saturn */
		8.34336671824457987e+00,
		4.12479856412430479e+00,
		-4.03523417114321381e-01,
		-2.76742510726862411e-03 * days_per_year,
		4.99852801234917238e-03 * days_per_year,
		2.30417297573763929e-05 * days_per_year,
		2.85885980666130812e-04 * solar_mass
	},

	{                               /* uranus */
		1.28943695621391310e+01,
		-1.51111514016986312e+01,
		-2.23307578892655734e-01,
		2.96460137564761618e-03 * days_per_year,
		2.37847173959480950e-03 * days_per_year,
		-2.96589568540237556e-05 * days_per_year,
		4.36624404335156298e-05 * solar_mass
	},

	{                               /* neptune */
		1.53796971148509165e+01,
		-2.59193146099879641e+01,
		1.79258772950371181e-01,
		2.68067772490389322e-03 * days_per_year,
		1.62824170038242295e-03 * days_per_year,
		-9.51592254519715870e-05 * days_per_year,
		5.15138902046611451e-05 * solar_mass
	}
};

const type DT{ 1e-2 };
const type RECIP_DT{ 1.0 / DT };

/*
* Rescale certain properties of bodies. That allows doing
* consequential advance()'s as if dt were equal to 1.0.
*
* When all advances done, rescale bodies back to obtain correct energy.
*/
template <typename T>
void scale_bodies(int nbodies, planet<T> *bodies, T scale) {
	//GPU
	for (int i = 0; i < nbodies; ++i) {
		bodies[i].mass *= scale*scale;
		bodies[i].vx *= scale;
		bodies[i].vy *= scale;
		bodies[i].vz *= scale;
	}
}

//template <typename T>
//void init_random_bodies(int nbodies, planet<T> *bodies) {
//
//	//GPU
//	for (int i = 0; i < nbodies; ++i) {
//		bodies[i].x = (T)rand() / RAND_MAX;
//		bodies[i].y = (T)rand() / RAND_MAX;
//		bodies[i].z = (T)rand() / RAND_MAX;
//		bodies[i].vx = (T)rand() / RAND_MAX;
//		bodies[i].vy = (T)rand() / RAND_MAX;
//		bodies[i].vz = (T)rand() / RAND_MAX;
//		bodies[i].mass = (T)rand() / RAND_MAX;
//	}
//}

template <typename T>
__global__ void init_random_bodies(int nBodies, planet<T> *bodies) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	hiprandState state;
	hiprand_init((unsigned long long)clock() + idx, 0, 0, &state);

	if (idx < nBodies){
		bodies[idx].x = hiprand_uniform_double(&state) + 0.0001;
		bodies[idx].y = hiprand_uniform_double(&state) + 0.0001;
		bodies[idx].z = hiprand_uniform_double(&state) + 0.0001;
		bodies[idx].vx = hiprand_uniform_double(&state) + 0.0001;
		bodies[idx].vy = hiprand_uniform_double(&state) + 0.0001;
		bodies[idx].vz = hiprand_uniform_double(&state) + 0.0001;
		bodies[idx].mass = hiprand_uniform_double(&state) + 0.0001;
	}
}

int main(int argc, char ** argv) {

	//CPU
	int niters = 1000, nbodies = 5;
	if (argc > 1) { niters = atoi(argv[1]); }
	if (argc > 2) { nbodies = atoi(argv[2]); }

	std::cout << "niters=" << niters << " nbodies=" << nbodies << '\n';

	planet<type> *bodies;
	if (argc == 1) {
		bodies = golden_bodies; // Check accuracy with 1000 solar system iterations
	}
	else {
		bodies = new planet<type>[nbodies];

		planet<type> *cudaBodies;
		hipMalloc((void**)&cudaBodies, nbodies * sizeof(planet<type>));

		hipMemcpy(cudaBodies, bodies, nbodies * sizeof(planet<type>), hipMemcpyHostToDevice);
		hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, init_random_bodies<type>, 0, nbodies);
		gridSize = (nbodies + blockSize - 1) / blockSize;
		init_random_bodies<<<gridSize, blockSize >>>(nbodies, cudaBodies);

		hipMemcpy(bodies, cudaBodies, nbodies * sizeof(planet<type>), hipMemcpyDeviceToHost);

		//init_random_bodies(nbodies, bodies); //Old Function
 	}

	auto t1 = std::chrono::steady_clock::now();
	offset_momentum(nbodies, bodies); //GPU
	type e1 = energy(nbodies, bodies); //GPU
	scale_bodies(nbodies, bodies, DT);

	for (int i = 1; i <= niters; ++i)  {
		advance(nbodies, bodies);
	}
	scale_bodies(nbodies, bodies, RECIP_DT);

	type e2 = energy(nbodies, bodies);
	auto t2 = std::chrono::steady_clock::now();
	auto diff = t2 - t1;

	std::cout << std::setprecision(9);
	std::cout << e1 << '\n' << e2 << '\n';
	std::cout << std::fixed << std::setprecision(3);
	std::cout << std::chrono::duration<double>(diff).count() << " seconds.\n";

	if (argc != 1) { delete[] bodies; }
	return 0;
}
